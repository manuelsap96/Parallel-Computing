#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <iostream>
#include <fstream>
#include <string>
#include "utils/des.h"

using namespace std;

#define IS_BIG_ENDIAN() (*(uint16_t *)"\0\xff"< 0x100)

//template structure to pass Thrust device vector to CUDA kernel
template <typename T>

struct KernelArray {
	T* _array;
	int _size;
};


//function to convert device vector to KernelArray structure
template <typename T>
KernelArray<T> convertToKernel(thrust::device_vector<T>& dVec) {
	KernelArray<T> kArray;
	kArray._array = thrust::raw_pointer_cast(&dVec[0]);
	kArray._size = (int) dVec.size();

	return kArray;
}



//function to convert a string to uint64
__device__ __host__ uint64_t str2uint64(const char* input){
	uint64_t output = 0;
	if (IS_BIG_ENDIAN()){
		for(int i = 0; i < 8; i++){
			output |= (uint64_t)input[i];
			if(i < 7)
				output <<= 8;
		}
	}
	else{
		for(int i = 7; i > -1; i--){
			output <<= 8;
			output |= (uint64_t)input[i];
		}
	}
	return output;
}




//kernel function
__global__ void decrypt_kernel(KernelArray<uint64_t> device_list, uint64_t u_salt, uint64_t crypt_pass, int *foundD, uint64_t *resultD) {

	uint64_t crypt;
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	//stride is the total number of threads
	int stride = blockDim.x * gridDim.x;

	while (i < device_list._size) {
        //password encryption from the list of passwords
		crypt = full_des_encode_block(device_list._array[i], u_salt);

		if (crypt_pass == crypt) {
             //password found
			*foundD = 1;
			*resultD = crypt;
			printf("Password matched at position %d\n", i+1);
			return;
		}
		i += stride;
	}
}


int main(void) {
	//in host_list there is the storage of all the passwords in the list
	thrust::host_vector<uint64_t> host_list(1);
	int block_size = 128;
	string salt = "F4";
	string password_target;
	string passwords_to_find[] = {"Manuel96","Paololep","carlo666"};
	//string passwords_to_find[] = {"freese93","feelin74","RaZeRis6","pan20139","cork1084","38993414","ybrf1721","vatino04","marioro9", "carlo666"};


	int* foundD;  //used in the device to confirm that the password_target was found
	uint64_t* resultD; //used in the device to storage the result

	//memory allocation
	hipMalloc((void**)&foundD, sizeof(int));
	hipMalloc((void**)&resultD, sizeof(uint64_t));
	hipMemset(foundD, 0, sizeof(int));
	hipMemset(resultD, 0, sizeof(uint64_t));

	int* foundH;  //used in the host
	uint64_t* resultH;  //used in the host

	//memory allocation
	foundH = (int *)malloc(sizeof(int));
	resultH = (uint64_t *)malloc(sizeof(uint64_t));

	int index = 0; //index used to resize the host list
	ifstream inputfile("password_dictionaryNew.txt");

	if (inputfile.is_open()) {
		string password;
		while (getline(inputfile, password)) {
			host_list[index] = str2uint64(password.c_str());
			index++;
			host_list.resize(index+1);
		}

		host_list.resize(index);
		inputfile.close();

	}
	else
		cout << "Error open file!" << endl;

	//block and grid dimension
	dim3 blockDim(block_size);
	dim3 gridDim(host_list.size()/blockDim.x + 1);

	 for(int i = 0; i<3; i++){
		password_target=passwords_to_find[i];
		for(int j = 0; j<3; j++){
			//returns the size of vector host_list
			std::cout << "Password list has size:  " << host_list.size() << std::endl;

			

			uint64_t u_salt = str2uint64(salt.c_str());
			uint64_t pass = str2uint64(password_target.c_str());
			uint64_t crypt_pass;
			//password_target encryption
			crypt_pass = full_des_encode_block(pass,u_salt);
			cout << "Password to find:" << password_target << endl;
			cout << "Password (crypted) to find: " << crypt_pass << endl;

			//transfer data to the device
			thrust::device_vector<uint64_t> device_list = host_list;

			clock_t time_start = clock();

			//threads running on the GPU
			decrypt_kernel<<<gridDim, blockDim>>>(convertToKernel(device_list), u_salt, crypt_pass, foundD, resultD );


			//save found and result to the host memory
			hipMemcpy(foundH, foundD, sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(resultH, resultD, sizeof(uint64_t), hipMemcpyDeviceToHost);


			if (*foundH == 1) {
				clock_t time_end = clock();
				float time_elapsed = (float)(time_end - time_start) / CLOCKS_PER_SEC;
				cout << "Password found!" << endl;
				cout << "Time elapsed in GPU: "<< time_elapsed * 1000 << " ms" << endl;
				cout << "The search result is : " << *resultH << endl << endl;
			}
			else
				cout << "Error! Password not found!" << endl;


			//data from device to host
			thrust::copy(device_list.begin(), device_list.end(), host_list.begin());

		}
	}
	return 0;
}



